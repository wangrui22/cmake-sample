#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>

__global__ void kernel_smooth(unsigned char *rgb, int w, int h, unsigned char *rgb_out) {
    unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < 1 || x > w-1 || y < 1 || y > h-3){
        return;
    }
    
    unsigned int x0 = x - 1;
    unsigned int y0 = y - 1;
    unsigned int x1 = x;
    unsigned int y1 = y;
    unsigned int x2 = x + 1;
    unsigned int y2 = y + 1;

    unsigned int i00 = y0*w + x0;
    unsigned int i01 = y1*w + x0;
    unsigned int i02 = y2*w + x0;
    unsigned int i10 = y0*w + x1;
    unsigned int i11 = y1*w + x1;
    unsigned int i12 = y2*w + x1;
    unsigned int i20 = y0*w + x2;
    unsigned int i21 = y1*w + x2;
    unsigned int i22 = y2*w + x2;

    int r = (int)rgb[3*i00] + (int)rgb[3*i01] + (int)rgb[3*i02] +
            (int)rgb[3*i10] + 2*(int)rgb[3*i11] + (int)rgb[3*i12] +
            (int)rgb[3*i20] + (int)rgb[3*i21] + (int)rgb[3*i22];
    r /= 10;

    int g = (int)rgb[3*i00+1] + (int)rgb[3*i01+1] + (int)rgb[3*i02+1] +
            (int)rgb[3*i10+1] + 2*(int)rgb[3*i11+1] + (int)rgb[3*i12+1] +
            (int)rgb[3*i20+1] + (int)rgb[3*i21+1] + (int)rgb[3*i22+1];
    g /= 10;

    int b = (int)rgb[3*i00+2] + (int)rgb[3*i01+2] + (int)rgb[3*i02+2] +
            (int)rgb[3*i10+2] + 2*(int)rgb[3*i11+2] + (int)rgb[3*i12+2] +
            (int)rgb[3*i20+2] + (int)rgb[3*i21+2] + (int)rgb[3*i22+2];
    b /= 10;
    
    rgb_out[i11*3] = r;
    rgb_out[i11*3+1] = g;
    rgb_out[i11*3+2] = b;
}

extern "C" 
hipError_t smooth(unsigned char *rgb, int w, int h, unsigned char *rgb_out) {
    dim3 block(16, 16, 1);
    dim3 grid(w / 16, h / 16, 1);

    kernel_smooth<<<grid, block>>>(rgb, w, h, rgb_out);
    

    return hipDeviceSynchronize();
}